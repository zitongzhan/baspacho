/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {
  hipDeviceProp_t dP;

  int rc = hipGetDeviceProperties(&dP, 0);
  if (rc != hipSuccess) {
    hipError_t error = hipGetLastError();
    printf("CUDA error: %s", hipGetErrorString(error));
    return rc; /* Failure */
  }
  printf("%d%d", dP.major, dP.minor);
  return 0;
}
